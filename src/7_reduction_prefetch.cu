#include "hip/hip_runtime.h"
#include "reduction.h"
#include <stdio.h>
#define BLOCK_SIZE 256 // must be power of 2
#define TN 8           // number of elements per thread
#define K 2            // whole data to seperate K parts

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/
// unroll warp reduction
__device__ void warpReduce4(volatile float* s_data, int localIdx, int current) {
    int index = current * BLOCK_SIZE + localIdx;
    s_data[index] += s_data[index + 32]; 
    s_data[index] += s_data[index + 16]; 
    s_data[index] += s_data[index + 8]; 
    s_data[index] += s_data[index + 4]; 
    s_data[index] += s_data[index + 2]; 
    s_data[index] += s_data[index + 1]; 
}

__device__ void blockReduce(volatile float* s_data, int localIdx, int current) {

    // do reduction in shared mem, interleave addressing
    int index = current * BLOCK_SIZE + localIdx;

    #pragma unroll
    for (unsigned int stride = blockDim.x/2; stride > 32; stride >>= 1) {
        if (localIdx < stride) {
            s_data[index] += s_data[index + stride];
        }
        __syncthreads();
    }

    // unroll warp reduction
    if (localIdx < 32) {
        warpReduce4(s_data, localIdx, current);
    }
}

__global__ void reduction_prefetech_kernel(float *d_output, const float *d_input, const int numElements) {
    unsigned int localIdx = threadIdx.x;
    unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // shared memory double buffering
    __shared__ float s_data[2 * BLOCK_SIZE];

    // Determine which shared memory bank to use for double buffering
    int current = 0;

    // seperate data to K parts
    int data_stride = numElements / K;
    int size = (data_stride + TN - 1) / TN;

    // Load the first data_stride into shared memory
    float4 input = {0.f, 0.f, 0.f, 0.f};

    #pragma unroll
    for (int i = globalIdx; i < size; i += blockDim.x * gridDim.x) {
        #pragma unroll
        for (int j = 0; j < TN; j+=4) {
            float4 temp = reinterpret_cast<const float4*>(&d_input[i * TN + j])[0];
            input.x += temp.x;
            input.y += temp.y;
            input.z += temp.z;
            input.w += temp.w;
        }
    }

    float inputSum = input.x + input.y + input.z + input.w;
    s_data[current * BLOCK_SIZE + localIdx] = inputSum;

    __syncthreads();

    // move data to next stride
    d_input += data_stride;

    // save the result of each data_stride
    float result[K] = {0.0f};
    int count = 0;

    #pragma unroll
    for (int bk = data_stride; bk < numElements; bk += data_stride, count++) {
        // Switch to the next shared memory bank for prefetching
        int next = 1 - current;

        // reset input
        input = {0.f, 0.f, 0.f, 0.f};

        #pragma unroll
        for (int i = globalIdx; i < size; i += blockDim.x * gridDim.x) {
            #pragma unroll
            for (int j = 0; j < TN; j+=4) {
                float4 temp = reinterpret_cast<const float4*>(&d_input[i * TN + j])[0];
                input.x += temp.x;
                input.y += temp.y;
                input.z += temp.z;
                input.w += temp.w;
            }
        }

        float inputSum = input.x + input.y + input.z + input.w;
        s_data[next * BLOCK_SIZE + localIdx] = inputSum;

        // move data to next stride
        d_input += data_stride;

        // perform block reduction
        blockReduce(s_data, localIdx, current);
        
        // save the result
        result[count] = s_data[current * BLOCK_SIZE];

        __syncthreads();

        // Move to the next data stride (next becomes current)
        current = next;
    }

    // Perform the last block reduction
    blockReduce(s_data, localIdx, current);
    result[count] = s_data[current * BLOCK_SIZE];

    // output 
    if (localIdx == 0) {
        int sum = 0;
        for (int i = 0; i < K; i++) {
            sum += result[i];
        }
        d_output[blockIdx.x] = sum;
    }
}

void reduction_prefetech(float *d_output, const float *d_input, const int numElements) {

    // calculate number of blocks
    int numSMs;
    int numBlocksPerSM;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, reduction_prefetech_kernel, BLOCK_SIZE, 0);
    int tileSize = BLOCK_SIZE * TN;
    int gridSize = min(numBlocksPerSM * numSMs, (numElements + (tileSize) - 1) / (tileSize));

    // launch kernel
    reduction_prefetech_kernel<<<gridSize, BLOCK_SIZE>>>(d_output, d_input, numElements);
    reduction_prefetech_kernel<<<1, BLOCK_SIZE>>>(d_output, d_output, gridSize);
}
